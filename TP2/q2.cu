
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecmabite( int *out, int *in, int threads, std::size_t size )
{
    auto tid_x = threadIdx.x;
    auto tid_b = blockIdx.x;
    out[ tid_x  + threads * tid_b] = in[ 2 * (tid_x  + threads * tid_b) ];
}

int main(int ac, char **av)
{
    if (ac < 2)
        return (-1);
    int len = atoi(av[1]);
    int * out_d = nullptr;
    int * in_d = nullptr;
    int thread_max = 0;
    int thread_x = 0;

    std::vector< int > out( len );
    std::vector< int > in( 2 * len );


    hipDeviceGetAttribute(&thread_max, hipDeviceAttributeMaxThreadsPerBlock, 0);
    if ((2 * len) / thread_max > 0)
        thread_x = 1024;
    else
        thread_x = thread_max;

    for( std::size_t i = 0 ; i < in.size() ; ++i )
    {
        in[ i ] = i;
    }

    hipMalloc( &out_d, out.size() * sizeof( int ) );
    hipMalloc( &in_d, in.size() * sizeof( int ) );
    hipMemcpy( in_d, in.data(), in.size() * sizeof( int ), hipMemcpyHostToDevice );

    vecmabite<<< (2 * len) / thread_max, thread_x  >>>( out_d, in_d, thread_max, out.size() );

    hipMemcpy( out.data(), out_d, out.size() * sizeof( int ), hipMemcpyDeviceToHost );

    for( auto const x: out )
    {
        std::cout << x << std::endl;
    }

    hipFree( out_d );
    hipFree( in_d );

    return 0;
}