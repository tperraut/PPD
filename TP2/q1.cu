
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int thread, block_x, block_y, block_z, grid_x, grid_y, grid_z, processor;

    thread = 0;
    block_x = 0;
    block_y = 0;
    block_z = 0;
    grid_x = 0;
    grid_y = 0;
    grid_z = 0;
    processor = 0;

    hipSetDevice(0);
    hipDeviceGetAttribute (&thread, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute (&block_x, hipDeviceAttributeMaxBlockDimX, 0);
    hipDeviceGetAttribute (&block_y, hipDeviceAttributeMaxBlockDimY, 0);
    hipDeviceGetAttribute (&block_z, hipDeviceAttributeMaxBlockDimZ, 0);
    hipDeviceGetAttribute (&grid_x, hipDeviceAttributeMaxGridDimX, 0);
    hipDeviceGetAttribute (&grid_y, hipDeviceAttributeMaxGridDimY, 0);
    hipDeviceGetAttribute (&grid_z, hipDeviceAttributeMaxGridDimZ, 0);
    hipDeviceGetAttribute (&processor, hipDeviceAttributeMultiprocessorCount, 0);

    std::cout << "thread : " << thread << std::endl;
    std::cout << "block: " << block_x << ", " << block_y << ", " << block_z << std::endl;
    std::cout << "grid: " << grid_x << ", " << grid_y << ", " << grid_z << std::endl;
    std::cout << "processor : " << processor << std::endl;

}