#include "hip/hip_runtime.h"
/**
 * Add 2 arrays of 100 elements on the device.
 */
#include <iostream>
#include <vector>
#include <algorithm>


__global__ void matadd( int ** v0, int ** v1, std::size_t size )
{
    auto tid_x = threadIdx.x;
    auto tid_y = threadIdx.y;
    v0[ tid_y ][tid_x] += v1[tid_y][ tid_x ];
}


int main(int ac, char **av)
{
    if (ac < 2)
        return (-1);
    int len = atoi(av[1]);
    std::vector<std::vector<int>>v0(len, std::vector<int>(len));
    std::vector<std::vector<int>>v1(len, std::vector<int>(len));
    std::vector< int > tmp( len );

    int ** v0_d = nullptr;
    int ** v1_d = nullptr;
    float elapsedTime;
    int k = 0;

    int thread_max = 0;
    int thread = 0;
    hipDeviceGetAttribute(&thread_max, hipDeviceAttributeMaxThreadsPerBlock, 0);
    if (len / thread_max > 0)
        thread = thread_max;
    else
        thread = len;
    //REMPLISSAGE
    for (std::vector<std::vector<int>>::iterator it = v0.begin() ; it != v0.end(); ++it)
    {
        for( std::size_t i = 0 ; i < (*it).size() ; ++i )
        {
            (*it)[ i ] = i;
        }
    }
    for (std::vector<std::vector<int>>::iterator it = v1.begin() ; it != v1.end(); ++it)
    {
        for( std::size_t i = 0 ; i < (*it).size() ; ++i )
        {
            (*it)[ i ] = i;
        }
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*
    //MALLOC matrice CUDA
    hipMalloc( &v0_d, v0.size() * sizeof( int * ) );
    for (std::vector<std::vector<int>>::iterator it = v0.begin() ; it != v0.end(); ++it)
    {
        hipMalloc( &(v0_d[k]), v0[k].size() * sizeof( int ) );
        ++k;
    }
    k = 0;
    hipMalloc( &v1_d, v1.size() * sizeof( int * ) );
    for (std::vector<std::vector<int>>::iterator it = v1.begin() ; it != v1.end(); ++it)
    {
        hipMalloc( &(v0_d[k]), v0[k].size() * sizeof( int ) );
        ++k;
    }
    //Remplissage matrice CUDA
    k = 0;
    for (std::vector<std::vector<int>>::iterator it = v0.begin() ; it != v0.end(); ++it)
    {
        hipMemcpy( v0_d[k], (v0[k]).data(), (v0[k]).size() * sizeof( int ), hipMemcpyHostToDevice );
        ++k;
    }
    k = 0;
    hipMalloc( &v1_d, v1.size() * sizeof( int * ) );
    for (std::vector<std::vector<int>>::iterator it = v1.begin() ; it != v1.end(); ++it)
    {
        hipMemcpy( v1_d[k], (v1[k]).data(), (v1[k]).size() * sizeof( int ), hipMemcpyHostToDevice );
        ++k;
    }

    hipEventRecord(start, 0);

    vecadd<<< 1, 1024 >>>( v0_d, v1_d, v0.size(), (v0[0]).size());
    k = 0;
    for (std::vector<std::vector<int>>::iterator it = v0.begin() ; it != v0.end(); ++it)
    {
        hipMemcpy( (v0[k]).data(), v0_d[k], (v0[k]).size() * sizeof( int ), hipMemcpyDeviceToHost );
        ++k;
    }
    */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    for( auto const x: v0 )
    {
        std::cout << x << std::endl;
    }
    std::cout << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree( v0_d );
    hipFree( v1_d );

    return 0;
}
