

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void ifpairmabite( int * v, std::size_t size )
{
	// Get the id of the thread ( 0 -> 99 ).
	auto tid = threadIdx.x;
	// Each thread fills a single element of the array. 
	if (!(v[tid] % 2))
		v[ tid ] *= 2;
}


int main()
{
	std::vector< int > v( 100 );

	int * v_d = nullptr;

	// Allocate an array an the device.
	for( std::size_t i = 0 ; i < v.size() ; ++i )
	{
		v[ i ] = i;
	}

	hipMalloc( &v_d, v.size() * sizeof( int ) );
	hipMemcpy( v_d, v.data(), v.size() * sizeof( int ), hipMemcpyHostToDevice );

	ifpairmabite<<< 1, 100 >>>( v_d, v.size() );

	hipMemcpy( v.data(), v_d, v.size() * sizeof( int ), hipMemcpyDeviceToHost );

	for( auto x: v )
	{
		std::cout << x << std::endl;
	}

	hipFree( v_d );

	return 0;
}
