/**
 * Add 2 arrays of 100 elements on the device.
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>


__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
	auto tid = threadIdx.x;
	v0[ tid ] += v1[ tid ];
}


int main()
{
	std::vector< int > v0( 1024 );
	std::vector< int > v1( 1024 );

	int * v0_d = nullptr;
	int * v1_d = nullptr;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for( std::size_t i = 0 ; i < v0.size() ; ++i )
	{
		v0[ i ] = v1[ i ] = i;
	}

	hipMalloc( &v0_d, v0.size() * sizeof( int ) );
	hipMalloc( &v1_d, v1.size() * sizeof( int ) );

	hipEventRecord(start, 0);
	hipMemcpy( v0_d, v0.data(), v0.size() * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );

	vecadd<<< 1, 1024 >>>( v0_d, v1_d, v0.size() );

	hipMemcpy( v0.data(), v0_d, v0.size() * sizeof( int ), hipMemcpyDeviceToHost );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	for( auto const x: v0 )
	{
		std::cout << x << std::endl;
	}
	std::cout << elapsedTime << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree( v0_d );
	hipFree( v1_d );

	return 0;
}
