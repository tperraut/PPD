
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecmabite( int *out, int *in, std::size_t size )
{
  auto tid = threadIdx.x;
  out[ tid ] = in[ 2 * tid ];
}

int main()
{
  std::vector< int > out( 50 );
  std::vector< int > in( 100 );
  
  int * out_d = nullptr;
  int * in_d = nullptr;

  for( std::size_t i = 0 ; i < in.size() ; ++i )
  {
    in[ i ] = i;
  }
  hipMalloc( &out_d, out.size() * sizeof( int ) );
  hipMalloc( &in_d, in.size() * sizeof( int ) );
  hipMemcpy( in_d, in.data(), in.size() * sizeof( int ), hipMemcpyHostToDevice );
  
  vecmabite<<< 1, 100 >>>( out_d, in_d, out.size() );

  hipMemcpy( out.data(), out_d, out.size() * sizeof( int ), hipMemcpyDeviceToHost );

  for( auto const x: out )
  {
    std::cout << x << std::endl;
  }

  hipFree( out_d );
  hipFree( in_d );

  return 0;
}
